/*
*   In His Exalted Name
*   Vector Addition - Sequential Code
*   Ahmad Siavashi, Email: siavashi@aut.ac.ir
*   21/05/2018
*/
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"

#define TILE_WIDTH 16

void fillVector(float * v, size_t n);
void printVector(float * v, size_t n);


__global__ void matrixMulCUDA(float *c,  float *a, float *b , int n){
  int k ;
 int row = blockIdx.y * blockDim.y + threadIdx.y;
 int col = blockIdx.x * blockDim.x + threadIdx.x;
        float c_val = 0;
        for (int k = 0 ; k < n ; k++){
            float a_elem = a[row * n + k];
            float b_elem = b[k*n + col];
            c_val += a_elem* b_elem;
        }
        c[row * n + col] = c_val;
}




int main()
{
    const int vectorSize = 1024 * 16;
    float a[vectorSize], b[vectorSize], c[vectorSize];
    
    fillVector(a, vectorSize);
    fillVector(b, vectorSize);
 
 printVector(a ,vectorSize );
 float *dev_a = 0;
float *dev_b = 0;
float *dev_c = 0;
hipError_t cudaStatus;



cudaStatus = hipSetDevice(0);
if (cudaStatus != hipSuccess) {
printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
}
 
 cudaStatus = hipMalloc((void**)&dev_c, vectorSize * sizeof(float));
if (cudaStatus != hipSuccess) {
printf("hipMalloc failed!");
}
 
 cudaStatus = hipMalloc((void**)&dev_a, vectorSize * sizeof(float));
if (cudaStatus != hipSuccess) {
printf("hipMalloc failed!");
}
cudaStatus = hipMalloc((void**)&dev_b, vectorSize * sizeof(float));
if (cudaStatus != hipSuccess) {
printf("hipMalloc failed!");
}

cudaStatus = hipMemcpy(dev_a, a, vectorSize * sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
printf("hipMemcpy failed!");
}
cudaStatus = hipMemcpy(dev_b, b, vectorSize * sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
printf("hipMemcpy failed!");
}
 dim3 DimGrid( 16, 16, 1);
dim3 DimBlock(8,8, 1);
matrixMulCUDA <<<DimGrid, DimBlock>>>(dev_c, dev_a, dev_b , 128 );
cudaStatus = hipGetLastError();
if (cudaStatus != hipSuccess) {
printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
}
cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n",
cudaStatus);
}
cudaStatus = hipMemcpy(c, dev_c, vectorSize * sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {
printf("hipMemcpy failed!");
}
 
hipFree(dev_c);
hipFree(dev_a);
hipFree(dev_b);
printVector(c, vectorSize);
return cudaStatus;

}


// Fills a vector with data
void fillVector(float * v, size_t n) {
    int i;
  //((float)rand()/RAND_MAX)*10;      
    for (i = 0; i < n; i++) {
        v[i] =  1.0f;                                //((float)rand()/RAND_MAX)*10;  
    }
}


// Prints a vector to the stdout.
void printVector(float * v, size_t n) {
    int i;
    printf("[-] Vector elements: ");
    for (i = 0; i < n; i++) {
        printf("%f, ", v[i]);
    }
    printf("\b\b  \n");
}

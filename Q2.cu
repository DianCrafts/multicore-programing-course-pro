/*
*   In His Exalted Name
*   Vector Addition - Sequential Code
*   Ahmad Siavashi, Email: siavashi@aut.ac.ir
*   21/05/2018
*/
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"

#define TILE_WIDTH 16

void fillVector(float * v, size_t n);
void printVector(float * v, size_t n);


__global__ void matrixMulCUDA(float *c,  float *a, float *b , int n){
  int k ;
 int start_row = threadIdx.y * TILE_WIDTH;
 int end_row = start_row + TILE_WIDTH;
 int start_col = threadIdx.x * TILE_WIDTH;
 int end_col = start_col + TILE_WIDTH;
for (int row = start_row ; row < end_row ; row++){
    for(int col = start_col ; col < end_col ; col++){
        float c_val = 0;
        for (int k = 0 ; k < n ; k++){
            float a_elem = a[row * n + k];
            float b_elem = b[k*n + col];
            c_val += a_elem* b_elem;
        }
        c[row * n + col] = c_val;
    }
}
}




int main()
{
    const int vectorSize = 1024 * 16 * 4;
    float a[vectorSize], b[vectorSize], c[vectorSize];
    
    fillVector(a, vectorSize);
    fillVector(b, vectorSize);
 
 printVector(a ,vectorSize );
 float *dev_a = 0;
float *dev_b = 0;
float *dev_c = 0;
hipError_t cudaStatus;
 hipEvent_t start, stop;
    float time;


cudaStatus = hipSetDevice(0);
if (cudaStatus != hipSuccess) {
printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
}
 
 cudaStatus = hipMalloc((void**)&dev_c, vectorSize * sizeof(float));
if (cudaStatus != hipSuccess) {
printf("hipMalloc failed!");
}
 
 cudaStatus = hipMalloc((void**)&dev_a, vectorSize * sizeof(float));
if (cudaStatus != hipSuccess) {
printf("hipMalloc failed!");
}
cudaStatus = hipMalloc((void**)&dev_b, vectorSize * sizeof(float));
if (cudaStatus != hipSuccess) {
printf("hipMalloc failed!");
}

cudaStatus = hipMemcpy(dev_a, a, vectorSize * sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
printf("hipMemcpy failed!");
}
cudaStatus = hipMemcpy(dev_b, b, vectorSize * sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
printf("hipMemcpy failed!");
}
// dim3 DimGrid( 1, 1, 1);
 
 hipEventCreate(&start);

    hipEventCreate(&stop);
    hipEventRecord(start, 0);
dim3 DimBlock(16,16, 1);
matrixMulCUDA <<<1, DimBlock>>>(dev_c, dev_a, dev_b , 256 );
 hipEventRecord(stop, 0);
 hipEventSynchronize(stop); //Block until the event is actually recorded
    hipEventElapsedTime(&time, start, stop);
 
 
    printf("Done in %f ms.\n", time);


cudaStatus = hipGetLastError();
if (cudaStatus != hipSuccess) {
printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
}
cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n",
cudaStatus);
}
cudaStatus = hipMemcpy(c, dev_c, vectorSize * sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {
printf("hipMemcpy failed!");
}
 
hipFree(dev_c);
hipFree(dev_a);
hipFree(dev_b);
printVector(c, vectorSize);
return cudaStatus;

}


// Fills a vector with data
void fillVector(float * v, size_t n) {
    int i;
  //((float)rand()/RAND_MAX)*10;      
    for (i = 0; i < n; i++) {
        v[i] =  1.0f;                                //((float)rand()/RAND_MAX)*10;  
    }
}


// Prints a vector to the stdout.
void printVector(float * v, size_t n) {
    int i;
    printf("[-] Vector elements: ");
    for (i = 0; i < n; i++) {
        printf("%f, ", v[i]);
    }
    printf("\b\b  \n");
}

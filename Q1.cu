/*
*   In His Exalted Name
*   Vector Addition - Sequential Code
*   Ahmad Siavashi, Email: siavashi@aut.ac.ir
*   21/05/2018
*/
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"


void fillVector(float * v, size_t n);
void printVector(float * v, size_t n);


__global__ void matrixMulCUDA(float *c,  float *a, float *b , int n){
    int k ;
 int row = threadIdx.y;
 int col = threadIdx.x;
 float sum = 0.0f ;
 for (k = 0 ; k < n ; ++k){
     sum += a[row* n +k] * b[k* n + col];
 }
 c[row* n + col] = sum;
}




int main()
{
    const int vectorSize = 1024;
    float a[vectorSize], b[vectorSize], c[vectorSize];
    
    fillVector(a, vectorSize);
    fillVector(b, vectorSize);
 
 printVector(a ,vectorSize );
 float *dev_a = 0;
float *dev_b = 0;
float *dev_c = 0;
hipError_t cudaStatus;



cudaStatus = hipSetDevice(0);
if (cudaStatus != hipSuccess) {
printf("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
}
 
 cudaStatus = hipMalloc((void**)&dev_c, vectorSize * sizeof(float));
if (cudaStatus != hipSuccess) {
printf("hipMalloc failed!");
}
 
 cudaStatus = hipMalloc((void**)&dev_a, vectorSize * sizeof(float));
if (cudaStatus != hipSuccess) {
printf("hipMalloc failed!");
}
cudaStatus = hipMalloc((void**)&dev_b, vectorSize * sizeof(float));
if (cudaStatus != hipSuccess) {
printf("hipMalloc failed!");
}

cudaStatus = hipMemcpy(dev_a, a, vectorSize * sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
printf("hipMemcpy failed!");
}
cudaStatus = hipMemcpy(dev_b, b, vectorSize * sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
printf("hipMemcpy failed!");
}
// dim3 DimGrid( 1, 1, 1);
dim3 DimBlock(32, 32, 1);
matrixMulCUDA <<<1, DimBlock>>>(dev_c, dev_a, dev_b , 32);
cudaStatus = hipGetLastError();
if (cudaStatus != hipSuccess) {
printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
}
cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n",
cudaStatus);
}
cudaStatus = hipMemcpy(c, dev_c, vectorSize * sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {
printf("hipMemcpy failed!");
}
 
hipFree(dev_c);
hipFree(dev_a);
hipFree(dev_b);
printVector(c, vectorSize);
return cudaStatus;

}


// Fills a vector with data
void fillVector(float * v, size_t n) {
    int i;
  //((float)rand()/RAND_MAX)*10;      
    for (i = 0; i < n; i++) {
        v[i] =  0.01f;                                //((float)rand()/RAND_MAX)*10;  
    }
}


// Prints a vector to the stdout.
void printVector(float * v, size_t n) {
    int i;
    printf("[-] Vector elements: ");
    for (i = 0; i < n; i++) {
        printf("%f, ", v[i]);
    }
    printf("\b\b  \n");
}
